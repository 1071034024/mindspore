#include "hip/hip_runtime.h"
/**
 * Copyright 2019 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <math.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "cross_entropy_impl.cuh"

template <typename T, typename S>
__global__ void CrossEntropyWithSparseKernel(const T *logits, const S *labels, const size_t batch_size,
                                             const size_t class_num, T *loss) {
  double total_loss = 0.0;
  T epsilon = 1e-6;
  for (size_t i = 0; i < batch_size; ++i) {
    T logit = logits[i * class_num + labels[i]];
    if (logit <= 0) {
      logit += epsilon;
    }
    total_loss += -logf(logit);
  }
  total_loss /= batch_size;
  loss[0] = static_cast<T>(total_loss);
  return;
}

template <typename T, typename S>
__global__ void CrossEntropyGradWithSparseKernel(const T *logits, const S *labels, const size_t batch_size,
                                                 const size_t class_num, T *grad) {
  for (size_t i = 0; i < batch_size; i++) {
    for (size_t j = blockIdx.x * blockDim.x + threadIdx.x; j < class_num; j += blockDim.x * gridDim.x) {
      if (labels[i] == j) {
        grad[i * class_num + j] = (logits[i * class_num + j] - 1) / batch_size;
      } else {
        grad[i * class_num + j] = logits[i * class_num + j] / batch_size;
      }
    }
  }
  return;
}

template <typename T, typename S>
__global__ void CrossEntropyWithoutSparseKernel(const T *logits, const S *labels, const size_t batch_size,
                                                const size_t class_num, T *losses) {
  T epsilon = 1e-6;
  for (size_t i = 0; i < batch_size; ++i) {
    T logit = 0.0;
    for (size_t j = 0; j < class_num; j++) {
      if (fabs(labels[i * class_num + j] - 1.0) <= 1e-8) {
        logit = logits[i * class_num + j];
        break;
      }
    }
    if (logit <= 0) {
      logit += epsilon;
    }
    losses[i] = -logf(logit);
  }
  return;
}

template <typename T, typename S>
__global__ void CrossEntropyGradWithoutSparseKernel(const T *logits, const S *labels, const size_t batch_size,
                                                    const size_t class_num, T *grad) {
  for (size_t i = 0; i < batch_size; i++) {
    for (size_t j = blockIdx.x * blockDim.x + threadIdx.x; j < class_num; j += blockDim.x * gridDim.x) {
      if (fabs(labels[i * class_num + j] - 1.0) <= 1e-8) {
        grad[i * class_num + j] = (logits[i * class_num + j] - 1) / batch_size;
      } else {
        grad[i * class_num + j] = logits[i * class_num + j] / batch_size;
      }
    }
  }
  return;
}

template <typename T, typename S>
void CrossEntropyWithSparse(const T *logits, const S *labels, const size_t batch_size, const size_t class_num, T *loss,
                            hipStream_t hip_stream) {
  CrossEntropyWithSparseKernel<<<1, 1, 0, hip_stream>>>(logits, labels, batch_size, class_num, loss);
  return;
}

template <typename T, typename S>
void CrossEntropyGradWithSparse(const T *logits, const S *labels, const size_t batch_size, const size_t class_num,
                                T *grad, hipStream_t hip_stream) {
  CrossEntropyGradWithSparseKernel<<<GET_BLOCKS(class_num), GET_THREADS, 0, hip_stream>>>(logits, labels, batch_size,
                                                                                           class_num, grad);
  return;
}

template <typename T, typename S>
void CrossEntropyWithoutSparse(const T *logits, const S *labels, const size_t batch_size, const size_t class_num,
                               T *losses, hipStream_t hip_stream) {
  CrossEntropyWithoutSparseKernel<<<1, 1, 0, hip_stream>>>(logits, labels, batch_size, class_num, losses);
  return;
}

template <typename T, typename S>
void CrossEntropyGradWithoutSparse(const T *logits, const S *labels, const size_t batch_size, const size_t class_num,
                                   T *grad, hipStream_t hip_stream) {
  CrossEntropyGradWithoutSparseKernel<<<GET_BLOCKS(class_num), GET_THREADS, 0, hip_stream>>>(
    logits, labels, batch_size, class_num, grad);
  return;
}

template void CrossEntropyWithSparse<float, int>(const float *logits, const int *labels, const size_t batch_size,
                                                 const size_t class_num, float *loss, hipStream_t hip_stream);
template void CrossEntropyWithSparse<float, int64_t>(const float *logits, const int64_t *labels,
                                                     const size_t batch_size, const size_t class_num, float *loss,
                                                     hipStream_t hip_stream);
template void CrossEntropyGradWithSparse<float, int>(const float *logits, const int *labels, const size_t batch_size,
                                                     const size_t class_num, float *grad, hipStream_t hip_stream);
template void CrossEntropyGradWithSparse<float, int64_t>(const float *logits, const int64_t *labels,
                                                         const size_t batch_size, const size_t class_num, float *grad,
                                                         hipStream_t hip_stream);
template void CrossEntropyWithoutSparse<float, float>(const float *logits, const float *labels, const size_t batch_size,
                                                      const size_t class_num, float *losses, hipStream_t hip_stream);
template void CrossEntropyGradWithoutSparse<float, float>(const float *logits, const float *labels,
                                                          const size_t batch_size, const size_t class_num, float *grad,
                                                          hipStream_t hip_stream);
